#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <string>

#define CHKMAL_ERROR	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
#define CHKMEMCPY_ERROR if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
#define CHKSYNC_ERROR	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error; }



// 300,000 indices; 2,000 total threads; 2 blocks: 1,000 threads per block;
// Each thread in charge of 150 contigeous indices
#define THREAD_BLOCK_SIZE 150	
#define NO_BLOCKS  2            
#define THREADS_PER_BLOCK 1000

// Helper function for using CUDA to add vectors in parallel.
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

// Histogram code
__global__ void threadedHistKernel(int *threadedHist, int *arr, const int blockSize, const int valRange, const int threadBlockSize)
{
	int val,
		bid = blockIdx.x,
		tid = threadIdx.x,
		pid = bid*blockSize + tid;  //positional ID

									// each thread takes info from its given info and increases the relevant position on the threadedHist
	for (int i = 0; i < threadBlockSize; i++)
	{
		val = arr[pid*threadBlockSize + i];
		threadedHist[valRange*pid + val]++;

	}
}

__global__ void sumThreadedResultsKernel(long *dev_hist, int *dev_threadedHist, const int valRange, const int Blocks)
{
	//e.g. tid from 0 to valRange-1, blocks = THREADS_PER_BLOCK * NO_BLOCKS
	int tid = threadIdx.x;

	for (int bl = 0; bl < Blocks; bl++)
	{
		dev_hist[tid] += dev_threadedHist[bl*valRange + tid];
	}
}

hipError_t histogramWithCuda(long* hist, const int* largeArr, const int arrSize, const int histSize)
{
	int  *dev_arr = 0;
	long *dev_hist = 0;
	int  *dev_threadedHist = 0;
	hipError_t cudaStatus;

	// memory init block
	{
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}

		// Allocate GPU buffers
		cudaStatus = hipMalloc((void**)&dev_arr, arrSize * sizeof(int)); CHKMAL_ERROR;
		cudaStatus = hipMalloc((void**)&dev_hist, histSize * sizeof(long)); CHKMAL_ERROR;
		cudaStatus = hipMalloc((void**)&dev_threadedHist, THREADS_PER_BLOCK * NO_BLOCKS * histSize * sizeof(int)); CHKMAL_ERROR;    // each thread gets a "private" 

		// Copy input / memSet (Host to Device)
		cudaStatus = hipMemcpy(dev_arr, largeArr, arrSize * sizeof(int), hipMemcpyHostToDevice); CHKMEMCPY_ERROR;
		cudaStatus = hipMemcpy(dev_hist, hist, histSize * sizeof(int), hipMemcpyHostToDevice); CHKMEMCPY_ERROR;

		cudaStatus = hipMemset((void*)dev_threadedHist, 0, THREADS_PER_BLOCK * NO_BLOCKS * histSize * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemset failed!\n");
			goto Error;
		}

	}

	// *** phase 1 ***
	// Launch a kernel on the GPU with one thread for every THREAD_BLOCK_SIZE elements.
	threadedHistKernel << <NO_BLOCKS, THREADS_PER_BLOCK >> >(dev_threadedHist, dev_arr, THREADS_PER_BLOCK, histSize, THREAD_BLOCK_SIZE);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "threadedHistKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	cudaStatus = hipDeviceSynchronize(); CHKSYNC_ERROR;

	// *** phase 2 ***
	sumThreadedResultsKernel << <1, histSize >> >(dev_hist, dev_threadedHist, histSize, THREADS_PER_BLOCK * NO_BLOCKS);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "sumThreadedResultsKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	cudaStatus = hipDeviceSynchronize(); CHKSYNC_ERROR;

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(hist, dev_hist, histSize * sizeof(int), hipMemcpyDeviceToHost); CHKMEMCPY_ERROR;

Error:
	hipFree(dev_arr);
	hipFree(dev_hist);
	hipFree(dev_threadedHist);

	return cudaStatus;

}







// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
